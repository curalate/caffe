#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------
#include <hipcub/hipcub.hpp>
#include <iomanip>

#include "caffe/layers/frcnn_proposal_layer.hpp"
#include "caffe/util/frcnn_utils.hpp"
#include "caffe/util/frcnn_helper.hpp"
#include "caffe/util/frcnn_param.hpp"
#include "caffe/util/frcnn_gpu_nms.hpp"

namespace caffe {

using std::vector;

__global__ void GetIndex(const int n,int *indices){
  CUDA_KERNEL_LOOP(index , n){
    indices[index] = index;
  }
}

template <typename Dtype>
__global__ void BBoxTransformInv(const int nthreads, const Dtype* const bottom_rpn_bbox,
                                 const int height, const int width, const int feat_stride,
                                 const int im_height, const int im_width,
                                 const int* sorted_indices, const float* anchors,
                                 float* const transform_bbox) {
  CUDA_KERNEL_LOOP(index , nthreads) {
    const int score_idx = sorted_indices[index];
    const int i = score_idx % width;  // width
    const int j = (score_idx % (width * height)) / width;  // height
    const int k = score_idx / (width * height); // channel
    float *box = transform_bbox + index * 4;
    box[0] = anchors[k * 4 + 0] + i * feat_stride;
    box[1] = anchors[k * 4 + 1] + j * feat_stride;
    box[2] = anchors[k * 4 + 2] + i * feat_stride;
    box[3] = anchors[k * 4 + 3] + j * feat_stride;
    const Dtype det[4] = { bottom_rpn_bbox[(k * 4 + 0) * height * width + j * width + i],
                           bottom_rpn_bbox[(k * 4 + 1) * height * width + j * width + i],
                           bottom_rpn_bbox[(k * 4 + 2) * height * width + j * width + i],
                           bottom_rpn_bbox[(k * 4 + 3) * height * width + j * width + i] };
    float src_w = box[2] - box[0] + 1;
    float src_h = box[3] - box[1] + 1;
    float src_ctr_x = box[0] + 0.5 * src_w;
    float src_ctr_y = box[1] + 0.5 * src_h;
    float pred_ctr_x = det[0] * src_w + src_ctr_x;
    float pred_ctr_y = det[1] * src_h + src_ctr_y;
    float pred_w = exp(det[2]) * src_w;
    float pred_h = exp(det[3]) * src_h;
    box[0] = pred_ctr_x - 0.5 * pred_w;
    box[1] = pred_ctr_y - 0.5 * pred_h;
    box[2] = pred_ctr_x + 0.5 * pred_w;
    box[3] = pred_ctr_y + 0.5 * pred_h;
    box[0] = max(0.0f, min(box[0], im_width - 1.0));
    box[1] = max(0.0f, min(box[1], im_height - 1.0));
    box[2] = max(0.0f, min(box[2], im_width - 1.0));
    box[3] = max(0.0f, min(box[3], im_height - 1.0));
  }
}

__global__ void SelectBox(const int nthreads, const float *box, float min_size,
                          int *flags) {
  CUDA_KERNEL_LOOP(index , nthreads) {
    if ((box[index * 4 + 2] - box[index * 4 + 0] < min_size) ||
        (box[index * 4 + 3] - box[index * 4 + 1] < min_size)) {
      flags[index] = 0;
    } else {
      flags[index] = 1;
    }
  }
}

template <typename Dtype>
__global__ void SelectBoxByIndices(const int nthreads, const float *in_box, int *selected_indices,
                          float *out_box, const Dtype *in_score, Dtype *out_score) {
  CUDA_KERNEL_LOOP(index , nthreads) {
    if ((index == 0 && selected_indices[index] == 1) ||
        (index > 0 && selected_indices[index] == selected_indices[index - 1] + 1)) {
      out_box[(selected_indices[index] - 1) * 4 + 0] = in_box[index * 4 + 0];
      out_box[(selected_indices[index] - 1) * 4 + 1] = in_box[index * 4 + 1];
      out_box[(selected_indices[index] - 1) * 4 + 2] = in_box[index * 4 + 2];
      out_box[(selected_indices[index] - 1) * 4 + 3] = in_box[index * 4 + 3];
      if (in_score!=NULL && out_score!=NULL) {
        out_score[selected_indices[index] - 1] = in_score[index];
      }
    }
  }
}

template <typename Dtype>
__global__ void SelectBoxAftNMS(const int nthreads, const float *in_box, int *keep_indices,
                          Dtype *top_data, const Dtype *in_score, Dtype* top_score) {
  CUDA_KERNEL_LOOP(index , nthreads) {
    top_data[index * 5] = 0;
    int keep_idx = keep_indices[index];
    for (int j = 1; j < 5; ++j) {
      top_data[index * 5 + j] = in_box[keep_idx * 4 + j - 1];
    }
    if (top_score != NULL && in_score != NULL) {
      top_score[index] = in_score[keep_idx];
    }
  }
}

template <typename Dtype>
void FrcnnProposalLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
    const vector<Blob<Dtype> *> &top) {
  Forward_cpu(bottom, top);
  return ;
}

template <typename Dtype>
void FrcnnProposalLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
    const vector<bool> &propagate_down, const vector<Blob<Dtype> *> &bottom) {
  for (int i = 0; i < propagate_down.size(); ++i) {
    if (propagate_down[i]) {
      NOT_IMPLEMENTED;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FrcnnProposalLayer);

} // namespace caffe
